/*
 * Copyright 1993-2015 NVIDIA Corporation.	All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * ! with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
	

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "matPrint.h"
#include "filter.h"
#include "config.h"

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif


int main(int argc, char **argv)
{
	int devID;
	hipDeviceProp_t props;

	// This will pick the best possible CUDA capable device
	devID = findCudaDevice(argc, (const char **)argv);

	//Get GPU information
	checkCudaErrors(hipGetDevice(&devID));
	checkCudaErrors(hipGetDeviceProperties(&props, devID));
	printf("Device %d: \"%s\" with Compute %d.%d capability\n",
	devID, props.name, props.major, props.minor);

	//Kernel configuration, where a two-dimensional grid and
	
	dim3 dimGrid(ROWS,COLS);// DO NOT TOUCH THIS!!!!!
	dim3 dimBlock(3,3);
	
	int *h_srcmat =(int*) malloc(sizeof(int)*DIM);
	int *h_dstmat =(int*) malloc(sizeof(int)*DIM);
	int* d_srcmat;
	int* d_dstmat;

	for(int i =0 ; i<ROWS;i++){
		for(int j=0 ; j<COLS;j++){
		*(h_srcmat + i*COLS + j) = i*COLS + j;
		}
	}
	printf("src matrix\n");
	matPrint(h_srcmat,ROWS,COLS);
	int h_f[F_DIM]={0};
	for (int i=0;i<F_DIM;i++)h_f[i]=1;
	//matPrint(h_f,F_COLS,F_COLS);
	hipMalloc((void**)&d_srcmat,sizeof(int)*DIM);
	hipMalloc((void**)&d_dstmat,sizeof(int)*DIM);
	
	int* d_f;	
	hipMalloc((void**)&d_f,sizeof(int)*DIM);
	
	hipMemcpy(d_srcmat,h_srcmat,sizeof(int)*DIM,hipMemcpyHostToDevice);
	hipMemcpy(d_f,h_f,sizeof(int)*F_DIM,hipMemcpyHostToDevice);


	applyFilter<<<dimGrid, dimBlock>>>(d_srcmat,d_f,d_dstmat);

	hipDeviceSynchronize();

	hipMemcpy(h_dstmat,d_dstmat,sizeof(int)*DIM,hipMemcpyDeviceToHost);
	printf("src matrix\n");
	matPrint(h_dstmat,ROWS,COLS);
	hipFree(d_srcmat);
	hipFree(d_dstmat);
	hipFree(d_f);
	free(h_srcmat);
	free(h_dstmat);
	
	return EXIT_SUCCESS;
}

