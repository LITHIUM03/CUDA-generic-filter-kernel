#include "hip/hip_runtime.h"
#include "config.h"
#include "filter.h"

/* all sizes are defined in config to avoid unneeded copying. src and dst are ROWS X COLS matrices. fil is F_COLS X F_COLS matrix. F_DIM = F_COLS^2.
 * each block writes a specific element of the dst matrix, namely dst[blockIdx.x][blockIdx.y]
 * BLACK_MARGIN is the number of rows / cols of the dst matrix that are not affected by the filter. those areas are zeroed.
 * BLACK_MARGIN is defind as F_COLS/2 (floor division). for a 3X3 like sobel this number is 3.
 */

__global__ void applyFilter(int* src, int* fil, int* dst)
{      
	int dst_idx = blockIdx.y /*how many rows of blocks to skip*/ * gridDim.x \
		      /*how many block are there in a row*/ + blockIdx.x /* inner row offset*/;
	
	if ((blockIdx.x < BLACK_MARGIN ||blockIdx.x > COLS-1 - BLACK_MARGIN)||
		(blockIdx.y < BLACK_MARGIN ||blockIdx.y > ROWS-1 - BLACK_MARGIN)){
//		printf(" I am now painting black pixel at [%d, %d]\n",blockIdx.x,blockIdx.y);
		dst[dst_idx] = 0 ;
		return;
	}

	__shared__ int cache;
	if(0 == threadIdx.x && 0 == threadIdx.y)
		cache = 0;
	__syncthreads();
	//dst_idx is the index in the filter with which the thread will multiply.
	int t_idx = threadIdx.y*blockDim.x + threadIdx.x;
	
	//srd_idx is the index in the input array to which the thread will write.
	int src_idx = ((threadIdx.y-(F_COLS/2))+blockIdx.y) * COLS + ((threadIdx.x-(F_COLS/2))+blockIdx.x);

	atomicAdd(&cache,fil[t_idx] * src[src_idx]);
	
	__syncthreads();

	if(!(threadIdx.x-F_COLS/2) && !(threadIdx.y-F_COLS/2))
	dst[src_idx] = cache;
}

